#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <optix_prime/optix_prime.h>
#include <optixu/optixu_matrix_namespace.h>

#include "structs.hpp"

using namespace optix;

__host__ __device__ float3 makeTarget(int x, int y, int w, int h, const Matrix4x4 &invProj)
{
	float4 r = invProj * make_float4(float(x) * 2 / w - 1, float(y) * 2 / h - 1, 0.5, 1);
	return make_float3(r / r.w);
}

__global__ void buildRaysKernel(ODRay *ptr, int w, int h, const float3 origin, const Matrix4x4 invProj)
{
	uint tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid >= w * h)
		return;
	ODRay r;
	r.origin = origin;
	r.direction = normalize(makeTarget(tid % w, tid / w, w, h, invProj) - origin);
	ptr[tid] = r;
}

extern "C" void buildRays(ODRay *ptr, int w, int h, const float3 &origin, const Matrix4x4 &invProj, int callOnDevice)
{
	if (callOnDevice)
	{
		int threads = w * h;
		buildRaysKernel<<<(threads + 1023) / 1024, 1024>>>(ptr, w, h, origin, invProj);
	}
	else
	{
		ODRay r;
		for (int j = 0; j < h; ++j)
		{
			for (int i = 0; i < w; ++i)
			{
				uint tid = i + j * w;
				r.origin = origin;
				r.direction = normalize(makeTarget(i, j, w, h, invProj) - origin);
				ptr[tid] = r;
			}
		}
	}
}
